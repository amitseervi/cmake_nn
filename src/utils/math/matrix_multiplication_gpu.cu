#include "hip/hip_runtime.h"
#include<types/dim.h>
#include<assert.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include<>

__global__ void _multiply_matrix_gpu_util(Vector2<int>* dim_a, Vector2<int>* dim_b, int* a, int* b, int* c) {
	int i = blockDim.y * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (j >= dim_b->y) {
		return;
	}
	if (i >= dim_a->x) {
		return;
	}
	int sum = 0;
	for (int k = 0; k < dim_a->y; k++) {
		sum += a[(i * dim_a->x) + k] * b[(k * dim_b->x) + j];
	}
	c[((i * dim_a->x) + j)] = sum;
}

void matrix_multiplication_gpu_kernel(Vector2<int> dim_a, Vector2<int> dim_b, int* a, int* b, int* c) {
	assert(dim_a.y == dim_b.x);
	Vector2<int>* device_dim_a;
	Vector2<int>* device_dim_b;
	int* device_a;
	int* device_b;
	int* device_c;
	hipMalloc(&device_dim_a, sizeof(Vector2<int>));
	hipMalloc(&device_dim_b, sizeof(Vector2<int>));
	hipMalloc(&device_a, sizeof(int) * dim_a.x * dim_a.y);
	hipMalloc(&device_b, sizeof(int) * dim_b.x * dim_b.y);
	hipMalloc(&device_c, sizeof(int) * dim_a.x * dim_b.y);
	hipMemcpy(device_dim_a, &dim_a, sizeof(Vector2<int>), hipMemcpyHostToDevice);
	hipMemcpy(device_dim_b, &dim_b, sizeof(Vector2<int>), hipMemcpyHostToDevice);
	hipMemcpy(device_a, a, sizeof(int) * dim_a.x * dim_a.y, hipMemcpyHostToDevice);
	hipMemcpy(device_b, b, sizeof(int) * dim_b.x * dim_b.y, hipMemcpyHostToDevice);
	dim3 threads_per_block(16, 16);
	dim3 num_of_blocks((dim_a.x + 15) / 16, (dim_b.y + 15) / 16);
	_multiply_matrix_gpu_util <<< num_of_blocks, threads_per_block >>> (device_dim_a, device_dim_b, device_a, device_b, device_c);
	hipFree(device_dim_a);
	hipFree(device_dim_b);
	hipFree(device_a);
	hipFree(device_b);
	hipMemcpy(c, device_c, sizeof(int) * dim_a.x * dim_b.y, hipMemcpyDeviceToHost);
	hipFree(device_c);
}
